#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <fstream>


#include "flamegpu/flame_api.h"
#include "flamegpu/runtime/flamegpu_api.h"
#include "flamegpu/io/factory.h"
#include "flamegpu/util/nvtx.h"

#define PRED_PREY_INTERACTION_RADIUS 0.1f
#define SAME_SPECIES_AVOIDANCE_RADIUS 0.1f
#define DELTA_TIME 0.1f
#define PRED_SPEED_ADVANTAGE 1.1f
#define GAIN_FROM_FOOD_PREDATOR 25
#define PRED_KILL_DISTANCE 0.05f



// Function definitions
void printPopulation(AgentPopulation &pop);

// FLAMEGPU_AGENT_FUNCTION(function_name, input_message_type, output_message_type)

// Predator functions

FLAMEGPU_AGENT_FUNCTION(pred_output_location, MsgNone, MsgBruteForce) {
    const float id = FLAMEGPU->getVariable<int>("id");
    const float x = FLAMEGPU->getVariable<float>("x");
    const float y = FLAMEGPU->getVariable<float>("y");
    FLAMEGPU->message_out.setVariable<int>("id", id); 
    FLAMEGPU->message_out.setVariable<float>("x", x); 
    FLAMEGPU->message_out.setVariable<float>("y", y); 

    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(pred_follow_prey, MsgBruteForce, MsgNone) {
    // Fetch the predator's position
    const float predator_x = FLAMEGPU->getVariable<float>("x");
    const float predator_y = FLAMEGPU->getVariable<float>("y"); 

    // Find the closest prey by iterating the prey_location messages
    float closest_prey_x;
    float closest_prey_y;
    float closest_prey_distance = PRED_PREY_INTERACTION_RADIUS;
    int is_a_prey_in_range = 0;

    for (const auto& msg : FLAMEGPU->message_in) {
	// Fetch prey location
	const float prey_x = msg.getVariable<float>("x");
	const float prey_y = msg.getVariable<float>("y");

	// Check if prey is within sight range of predator
	const float dx = prey_x - predator_x;
	const float dy = prey_y - predator_y;
	const float distance = sqrt(dx*dx + dy*dy);

	// TODO: Move magic constant to env variable
	if (distance < 0.1f) {
	    closest_prey_x = prey_x;
	    closest_prey_y = prey_y;
	    closest_prey_distance = distance;
	    is_a_prey_in_range = 1;	
	} 
    }
    
    // If there was a prey in range, steer the predator towards it
    if (is_a_prey_in_range) {
	const float steer_x = closest_prey_x - predator_x;
	const float steer_y = closest_prey_y - predator_y;
	FLAMEGPU->setVariable<float>("steer_x", steer_x);	
	FLAMEGPU->setVariable<float>("steer_y", steer_y);	
    }

    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(pred_avoid, MsgBruteForce, MsgNone) {
    // Fetch this predator's position
    const float predator_x = FLAMEGPU->getVariable<float>("x");
    const float predator_y = FLAMEGPU->getVariable<float>("y"); 
    float avoid_velocity_x = 0.0f;
    float avoid_velocity_y = 0.0f;

    // Add a steering factor away from each other predator. Strength increases with closeness.
    for (const auto& msg : FLAMEGPU->message_in) {
	// Fetch location of other predator
	const float other_predator_x = msg.getVariable<float>("x");
	const float other_predator_y = msg.getVariable<float>("y");

	// Check if the two predators are within interaction radius
	const float dx = predator_x - other_predator_x;
	const float dy = predator_y - other_predator_y;
	const float distance = sqrt(dx*dx + dy*dy);

	// TODO: Original implementation tests id to remove self-avoidance. However, dx, dy == 0 so no need?
	// TODO: Move magic constant to env variable
	if (distance < SAME_SPECIES_AVOIDANCE_RADIUS) {
	    avoid_velocity_x += (SAME_SPECIES_AVOIDANCE_RADIUS / distance) * dx;
	    avoid_velocity_y += (SAME_SPECIES_AVOIDANCE_RADIUS / distance) * dy;
	} 
    }
    
    FLAMEGPU->setVariable<float>("steer_x", avoid_velocity_x);
    FLAMEGPU->setVariable<float>("steer_y", avoid_velocity_y);

    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(pred_move, MsgNone, MsgNone) {
    float predator_x = FLAMEGPU->getVariable<float>("x");
    float predator_y = FLAMEGPU->getVariable<float>("y");
    float predator_vx = FLAMEGPU->getVariable<float>("vx");
    float predator_vy = FLAMEGPU->getVariable<float>("vy");
    const float predator_fx = FLAMEGPU->getVariable<float>("steer_x");
    const float predator_fy = FLAMEGPU->getVariable<float>("steer_y");
    const float predator_life = FLAMEGPU->getVariable<int>("life");

    // TODO: There isn't any time scaling?
    // Integrate steering forces
    predator_vx += predator_fx;
    predator_vy += predator_fy;

    // TODO: Cap velocity

    // Integrate velocity
    predator_x += predator_vx * DELTA_TIME * PRED_SPEED_ADVANTAGE;
    predator_y += predator_vy * DELTA_TIME * PRED_SPEED_ADVANTAGE;

    // TODO: Bound the position within the environment

    // Update agent state
    FLAMEGPU->setVariable<float>("x", predator_x);
    FLAMEGPU->setVariable<float>("y", predator_y);
    FLAMEGPU->setVariable<float>("vx", predator_vx);
    FLAMEGPU->setVariable<float>("vy", predator_vy);

    // Reduce life by one unit of energy
    FLAMEGPU->setVariable<int>("life", predator_life -1);

    return ALIVE; 
}

FLAMEGPU_AGENT_FUNCTION(pred_eat_or_starve, MsgBruteForce, MsgNone) {
    const int predator_id = FLAMEGPU->getVariable<int>("id");
    int predator_life = FLAMEGPU->getVariable<int>("life");
    int isDead = 0;

    // Iterate prey_eaten messages to see if this predator ate a prey
    for (const auto& msg : FLAMEGPU->message_in) {
	if (msg.getVariable<int>("pred_id") == predator_id) {
	    predator_life += GAIN_FROM_FOOD_PREDATOR;
        }	    	
    }

    // Update agent state
    FLAMEGPU->setVariable<int>("life", predator_life);

    // Did the predator starve?
    if (predator_life < 1) {
        isDead = 1;
    }

    return isDead ? DEAD : ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(pred_reproduction, MsgNone, MsgNone) {
    // TODO: NOT YET IMPLEMENTED
    return ALIVE;
}

// Prey functions

FLAMEGPU_AGENT_FUNCTION(prey_output_location, MsgNone, MsgBruteForce) {
    const float id = FLAMEGPU->getVariable<int>("id");
    const float x = FLAMEGPU->getVariable<float>("x");
    const float y = FLAMEGPU->getVariable<float>("y");
    FLAMEGPU->message_out.setVariable<int>("id", id); 
    FLAMEGPU->message_out.setVariable<float>("x", x); 
    FLAMEGPU->message_out.setVariable<float>("y", y); 

    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_avoid_pred, MsgBruteForce, MsgNone) {
    // Fetch this prey's position
    const float prey_x = FLAMEGPU->getVariable<float>("x");
    const float prey_y = FLAMEGPU->getVariable<float>("y"); 
    float avoid_velocity_x = 0.0f;
    float avoid_velocity_y = 0.0f;

    // Add a steering factor away from each predator. Strength increases with closeness.
    for (const auto& msg : FLAMEGPU->message_in) {
	// Fetch location of predator
	const float predator_x = msg.getVariable<float>("x");
	const float predator_y = msg.getVariable<float>("y");

	// Check if the two predators are within interaction radius
	const float dx = prey_x - predator_x;
	const float dy = prey_y - predator_y;
	const float distance = sqrt(dx*dx + dy*dy);

	if (distance < PRED_PREY_INTERACTION_RADIUS) {
	    // Steer the prey away from the predator
	    avoid_velocity_x += (PRED_PREY_INTERACTION_RADIUS / distance) * dx;
	    avoid_velocity_y += (PRED_PREY_INTERACTION_RADIUS / distance) * dy;
	} 
    }
   
    // Update agent state 
    FLAMEGPU->setVariable<float>("steer_x", avoid_velocity_x);
    FLAMEGPU->setVariable<float>("steer_y", avoid_velocity_y);

    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_flock, MsgBruteForce, MsgNone) {
    // TODO: NOT YET IMPLEMENTED

    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_move, MsgNone, MsgNone) {
    float prey_x = FLAMEGPU->getVariable<float>("x");
    float prey_y = FLAMEGPU->getVariable<float>("y");
    float prey_vx = FLAMEGPU->getVariable<float>("vx");
    float prey_vy = FLAMEGPU->getVariable<float>("vy");
    const float prey_fx = FLAMEGPU->getVariable<float>("steer_x");
    const float prey_fy = FLAMEGPU->getVariable<float>("steer_y");
    const float prey_life = FLAMEGPU->getVariable<int>("life");

    // TODO: There isn't any time scaling?
    // Integrate steering forces
    prey_vx += prey_fx;
    prey_vy += prey_fy;

    // TODO: Cap velocity

    // Integrate velocity
    prey_x += prey_vx * DELTA_TIME; 
    prey_y += prey_vy * DELTA_TIME;

    // TODO: Bound the position within the environment

    // Update agent state
    FLAMEGPU->setVariable<float>("x", prey_x);
    FLAMEGPU->setVariable<float>("y", prey_y);
    FLAMEGPU->setVariable<float>("vx", prey_vx);
    FLAMEGPU->setVariable<float>("vy", prey_vy);

    // Reduce life by one unit of energy
    FLAMEGPU->setVariable<int>("life", prey_life -1);

    return ALIVE; 
}

FLAMEGPU_AGENT_FUNCTION(prey_eaten, MsgBruteForce, MsgBruteForce) {
    int eaten = 0;
    int predator_id = -1;
    float closest_pred = PRED_KILL_DISTANCE;
    const float prey_x = FLAMEGPU->getVariable<float>("x");
    const float prey_y = FLAMEGPU->getVariable<float>("y");

    // Iterate predator_location messages to find the closest predator
    for (const auto& msg : FLAMEGPU->message_in) {
        // Fetch location of predator
	const float predator_x = msg.getVariable<float>("x");
	const float predator_y = msg.getVariable<float>("y");

	// Check if the two predators are within interaction radius
	const float dx = prey_x - predator_x;
	const float dy = prey_y - predator_y;
	const float distance = sqrt(dx*dx + dy*dy);

	if (distance < closest_pred) {
	    predator_id = msg.getVariable<int>("id");
	    closest_pred= distance;
            eaten = 1;
	}
    }

    if (eaten)
	// TODO: Output prey_eaten message
	int a = 0;
	

    return eaten ? DEAD : ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_eat_or_starve, MsgBruteForce, MsgNone) {
    int isDead = 0;

    // Exercise 3.3 : TODO: Describe exercise 

    return isDead ? DEAD : ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(prey_reproduction, MsgNone, MsgNone) {
    // TODO: NOT YET IMPLEMENTED
    return ALIVE;
}

// Grass functions
FLAMEGPU_AGENT_FUNCTION(grass_output_location, MsgNone, MsgBruteForce) {
    // Exercise 3.1 : Set the variables for the grass_location message

    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(grass_eaten, MsgBruteForce, MsgNone) {
    // Exercise 3.2 : TODO: Describe exercise

    return ALIVE;
}

FLAMEGPU_AGENT_FUNCTION(grass_growth, MsgNone, MsgNone) {
    // Exercise 3.4 : TODO: Describe exercise 

    return ALIVE;
}


// Model definition


int main(int argc, const char ** argv) {
    NVTX_RANGE("main");
    NVTX_PUSH("ModelDescription");
    ModelDescription model("Tutorial_PredatorPrey_Example");

    /**
     * MESSAGE DEFINITIONS
     */

    {   // Grass location message
        MsgBruteForce::Description &message = model.newMessage("grass_location_message");
        message.newVariable<int>("id");
	message.newVariable<float>("x");
	message.newVariable<float>("y");
    }
    {   // Predator location message
        MsgBruteForce::Description &message = model.newMessage("predator_location_message");
        message.newVariable<int>("id");
	message.newVariable<float>("x");
	message.newVariable<float>("y");
    }
    {   // Prey location message
        MsgBruteForce::Description &message = model.newMessage("prey_location_message");
        message.newVariable<int>("id");
	message.newVariable<float>("x");
	message.newVariable<float>("y");
    }
    {   // Grass eaten message
        MsgBruteForce::Description &message = model.newMessage("grass_eaten_message");
        message.newVariable<int>("id");
        message.newVariable<int>("prey_id");
    }
    {   // Prey eaten message
        MsgBruteForce::Description &message = model.newMessage("prey_eaten_message");
        message.newVariable<int>("id");
        message.newVariable<int>("pred_id");
    }


    /**
     * AGENT DEFINITIONS
     */

    {   // Prey agent
        AgentDescription &agent = model.newAgent("prey");
        agent.newVariable<int>("id");
	agent.newVariable<float>("x");
	agent.newVariable<float>("y");
	agent.newVariable<float>("vx");
	agent.newVariable<float>("vy");
	agent.newVariable<float>("steer_x");
	agent.newVariable<float>("steer_y");
        agent.newVariable<int>("life");
        agent.newVariable<float>("type")
;
        agent.newFunction("prey_output_location", prey_output_location).setMessageOutput("prey_location_message");
        agent.newFunction("prey_avoid_pred", prey_avoid_pred).setMessageInput("predator_location_message");
        agent.newFunction("prey_flock", prey_flock).setMessageInput("prey_location_message");
        agent.newFunction("prey_move", prey_move);
	// TODO: Add optional prey_eaten output message for prey_eaten
        auto& function = agent.newFunction("prey_eaten", prey_eaten);
	function.setMessageInput("predator_location_message");
	function.setMessageOutput("prey_eaten_message");
        agent.newFunction("prey_eat_or_starve", prey_eat_or_starve).setMessageInput("grass_eaten_message");
	// TODO: Output new agent??
        agent.newFunction("prey_reproduction", prey_reproduction);
    }

    {   // Predator agent
        AgentDescription &agent = model.newAgent("predator");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
	agent.newVariable<float>("y");
	agent.newVariable<float>("vx");
	agent.newVariable<float>("vy");
	agent.newVariable<float>("steer_x");
	agent.newVariable<float>("steer_y");
        agent.newVariable<int>("life");
        agent.newVariable<float>("type");

        agent.newFunction("pred_output_location", pred_output_location).setMessageOutput("predator_location_message");
        agent.newFunction("pred_follow_prey", pred_follow_prey).setMessageInput("prey_location_message");
        agent.newFunction("pred_avoid", pred_avoid).setMessageInput("predator_location_message");
        agent.newFunction("pred_move", pred_move);
        agent.newFunction("pred_eat_or_starve", pred_eat_or_starve).setMessageInput("prey_eaten_message");
        agent.newFunction("pred_reproduction", pred_reproduction);
    }

    {   // Grass agent
	AgentDescription &agent = model.newAgent("grass");
        agent.newVariable<int>("id");
        agent.newVariable<float>("x");
	agent.newVariable<float>("y");
	agent.newVariable<int>("dead_cycles");
        agent.newVariable<int>("available");

	agent.newFunction("grass_output_location", grass_output_location).setMessageOutput("grass_location_message");
	// TODO: Add optional grass_eaten message
	agent.newFunction("grass_eaten", grass_eaten).setMessageInput("prey_location_message");
	agent.newFunction("grass_growth", grass_growth);
	
    }

   /**
     * ENVIRONMENT VARIABLES 
     */
    {
        EnvironmentDescription &env = model.Environment();
        env.add("REPRODUCE_PREY_PROB", 0.05f);
	env.add("REPRODUCE_PREDATOR_PROB", 0.03f);
	env.add("GAIN_FROM_FOOD_PREDATOR", 75);
	env.add("GAIN_FROM_FOOD_PREY", 50);
	env.add("GRASS_REGROW_CYCLES", 100);
    }

    /**
     * Control flow
     */
    {   // Layer #1
        LayerDescription &layer = model.newLayer();
        layer.addAgentFunction(prey_output_location);
        layer.addAgentFunction(pred_output_location);
        layer.addAgentFunction(grass_output_location);
    }
   // {   // Layer #2
   //     LayerDescription &layer = model.newLayer();
   //     layer.addAgentFunction(pred_follow_prey);
   //     layer.addAgentFunction(prey_avoid_pred);
   // }
   // {   // Layer #3
   //     LayerDescription &layer = model.newLayer();
   //     layer.addAgentFunction(prey_flock);
   //     layer.addAgentFunction(pred_avoid);
   // }
   // {   // Layer #4
   //     LayerDescription &layer = model.newLayer();
   //     layer.addAgentFunction(prey_move);
   //     layer.addAgentFunction(pred_move);
   // }
   // {   // Layer #5
   //     LayerDescription &layer = model.newLayer();
   //     layer.addAgentFunction(grass_eaten);
   //     layer.addAgentFunction(prey_eaten);
   // }
   // {   // Layer #6
   //     LayerDescription &layer = model.newLayer();
   //     layer.addAgentFunction(prey_eat_or_starve);
   //     layer.addAgentFunction(pred_eat_or_starve);
   // }
   // {   // Layer #7
   //     LayerDescription &layer = model.newLayer();
   //     layer.addAgentFunction(pred_reproduction);
   //     layer.addAgentFunction(prey_reproduction);
   //     layer.addAgentFunction(grass_growth);
   // }
    NVTX_POP();

    /**
     * Create Model Runner
     */
    NVTX_PUSH("CUDAAgentModel creation");
    CUDAAgentModel cuda_model(model);
    NVTX_POP();

    /**
     * Initialisation
     */
    cuda_model.initialise(argc, argv);
    if (cuda_model.getSimulationConfig().xml_input_file.empty()) {
	printf("XML Input was empty!");
    }


    /**
     * Execution
     */
    printf("Model initialised, beginning simulation...\n");
    printf("Step counter: %d\n Simulation Steps: %d", cuda_model.getStepCounter(), cuda_model.getSimulationConfig().steps);
     
    while (cuda_model.getStepCounter() < cuda_model.getSimulationConfig().steps && cuda_model.step()) {
	printf("On step %d", cuda_model.getStepCounter());
        //   cuda_model.getPopulationData(cell_pop);
        //  printPopulation(cell_pop);
        // getchar();
    }

   /**
    * Export Pop
    */
   // TODO
   return 0;
}

